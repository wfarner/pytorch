#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/native/cuda/Resize.cuh>
#include <ATen/native/ResizeCommon.h>

namespace at {
namespace native {

Tensor& resize_cuda_(
    Tensor& self,
    IntArrayRef size,
    c10::optional<MemoryFormat> optional_memory_format) {
  auto memory_format =
      optional_memory_format.value_or(MemoryFormat::Contiguous);
  TORCH_CHECK(
      memory_format != MemoryFormat::Preserve,
      "Unsupported memory format",
      memory_format);
#ifdef BUILD_NAMEDTENSOR
  if (self.has_names()) {
    return resize_named_tensor_(self, size, memory_format);
  }
#endif
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  self_->maybe_zero_dim(size.size() == 0);
  self_->empty_tensor_restride(memory_format);
  return self;
}
} // namespace native
} // namespace at
